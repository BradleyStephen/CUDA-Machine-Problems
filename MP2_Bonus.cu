#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <hip/hip_runtime.h>
#include <chrono>

#define NUM_ITER 10
#define TOLERANCE 1e-3f

#define TILE_ROW 12
#define TILE_COL 18
#define TILE_COMMON 18

// Revised tiled matrix multiplication kernel with boundary checks.
// Computes: P = M * N, where M is A_rows x A_cols and N is A_cols x B_cols.
__global__
void tiledMatMulKernelBoundary(const float* __restrict__ M,
    const float* __restrict__ N,
    float* __restrict__ P,
    int A_rows, int A_cols, int B_cols)
{
    extern __shared__ float shared[];
    float* tileM = shared;                      // TILE_ROW x TILE_COMMON
    float* tileN = shared + TILE_ROW * TILE_COMMON;  // TILE_COMMON x TILE_COL

    int row = blockIdx.y * TILE_ROW + threadIdx.y;
    int col = blockIdx.x * TILE_COL + threadIdx.x;
    float Pvalue = 0.0f;

    int numTiles = (A_cols + TILE_COMMON - 1) / TILE_COMMON;
    for (int t = 0; t < numTiles; t++)
    {
        int m_col = t * TILE_COMMON + threadIdx.x;
        if (row < A_rows && m_col < A_cols)
            tileM[threadIdx.y * TILE_COMMON + threadIdx.x] = M[row * A_cols + m_col];
        else
            tileM[threadIdx.y * TILE_COMMON + threadIdx.x] = 0.0f;

        for (int i = threadIdx.y; i < TILE_COMMON; i += TILE_ROW)
        {
            int n_row = t * TILE_COMMON + i;
            if (n_row < A_cols && col < B_cols)
                tileN[i * TILE_COL + threadIdx.x] = N[n_row * B_cols + col];
            else
                tileN[i * TILE_COL + threadIdx.x] = 0.0f;
        }

        __syncthreads();

        for (int k = 0; k < TILE_COMMON; k++)
            Pvalue += tileM[threadIdx.y * TILE_COMMON + k] * tileN[k * TILE_COL + threadIdx.x];

        __syncthreads();
    }

    if (row < A_rows && col < B_cols)
        P[row * B_cols + col] = Pvalue;
}

void cpuMatrixMulBoundary(const float* M, const float* N, float* P,
    int A_rows, int A_cols, int B_cols)
{
    for (int i = 0; i < A_rows; i++)
    {
        for (int j = 0; j < B_cols; j++)
        {
            float sum = 0.0f;
            for (int k = 0; k < A_cols; k++)
                sum += M[i * A_cols + k] * N[k * B_cols + j];
            P[i * B_cols + j] = sum;
        }
    }
}

bool compareResultsBoundary(const float* ref, const float* gpu, int size, float tolerance = TOLERANCE)
{
    for (int i = 0; i < size; i++)
    {
        if (fabs(ref[i] - gpu[i]) > tolerance)
            return false;
    }
    return true;
}

int main()
{
    FILE* fp = fopen("MP2Bonus.csv", "w");
    if (!fp)
    {
        printf("Error opening MP2Bonus.csv for writing.\n");
        return -1;
    }
    // CSV header: Test,A_rows,A_cols,B_cols,CPUTimeMs,GPUKernelAvgMs,GPUKernelStdMs,Result
    fprintf(fp, "Test,A_rows,A_cols,B_cols,CPUTimeMs,GPUKernelAvgMs,GPUKernelStdMs,Result\n");

    // Test cases:
    // Test 1: M: 750 x 800, N: 800 x 850 => P: 750 x 850
    // Test 2: M: 2000 x 1750, N: 1750 x 1900 => P: 2000 x 1900
    struct TestCase {
        int A_rows, A_cols, B_cols;
    } tests[2] = { {750, 800, 850}, {2000, 1750, 1900} };

    for (int test = 0; test < 2; test++)
    {
        int A_rows = tests[test].A_rows;
        int A_cols = tests[test].A_cols;
        int B_cols = tests[test].B_cols;
        size_t sizeM = A_rows * A_cols * sizeof(float);
        size_t sizeN = A_cols * B_cols * sizeof(float);
        size_t sizeP = A_rows * B_cols * sizeof(float);

        printf("Test %d: M: %d x %d, N: %d x %d, P: %d x %d\n", test + 1, A_rows, A_cols, A_cols, B_cols, A_rows, B_cols);

        float* h_M = (float*)malloc(sizeM);
        float* h_N = (float*)malloc(sizeN);
        float* h_P = (float*)malloc(sizeP);
        float* h_ref = (float*)malloc(sizeP);

        srand(0);
        for (int i = 0; i < A_rows * A_cols; i++)
            h_M[i] = static_cast<float>(rand() % 100) / 10.0f;
        for (int i = 0; i < A_cols * B_cols; i++)
            h_N[i] = static_cast<float>(rand() % 100) / 10.0f;

        // CPU reference multiplication timing
        auto cpuStart = std::chrono::high_resolution_clock::now();
        cpuMatrixMulBoundary(h_M, h_N, h_ref, A_rows, A_cols, B_cols);
        auto cpuEnd = std::chrono::high_resolution_clock::now();
        double cpuTime = std::chrono::duration<double, std::milli>(cpuEnd - cpuStart).count();
        printf("Test %d: CPU Reference Time: %.5f ms\n", test + 1, cpuTime);

        float* d_M, * d_N, * d_P;
        hipMalloc((void**)&d_M, sizeM);
        hipMalloc((void**)&d_N, sizeN);
        hipMalloc((void**)&d_P, sizeP);

        hipMemcpy(d_M, h_M, sizeM, hipMemcpyHostToDevice);
        hipMemcpy(d_N, h_N, sizeN, hipMemcpyHostToDevice);

        dim3 dimBlock(TILE_COL, TILE_ROW);  // 18 x 12 threads per block
        dim3 dimGrid((B_cols + TILE_COL - 1) / TILE_COL, (A_rows + TILE_ROW - 1) / TILE_ROW);
        size_t sharedMemSize = (TILE_ROW * TILE_COMMON + TILE_COMMON * TILE_COL) * sizeof(float);

        double sumKernel = 0.0, sumSqKernel = 0.0;
        for (int iter = 0; iter < NUM_ITER; iter++)
        {
            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start);
            tiledMatMulKernelBoundary << <dimGrid, dimBlock, sharedMemSize >> > (d_M, d_N, d_P, A_rows, A_cols, B_cols);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            float kernelTime = 0.0f;
            hipEventElapsedTime(&kernelTime, start, stop);
            hipEventDestroy(start);
            hipEventDestroy(stop);
            sumKernel += kernelTime;
            sumSqKernel += kernelTime * kernelTime;
        }
        double avgKernel = sumKernel / NUM_ITER;
        double stdKernel = sqrt((sumSqKernel / NUM_ITER) - (avgKernel * avgKernel));
        printf("Test %d: GPU Kernel Time: Avg = %.5f ms, Std = %.5f ms\n", test + 1, avgKernel, stdKernel);

        hipMemcpy(h_P, d_P, sizeP, hipMemcpyDeviceToHost);

        bool correct = compareResultsBoundary(h_ref, h_P, A_rows * B_cols);
        printf("Test %d: Result %s\n", test + 1, correct ? "PASSED" : "FAILED");
        printf("------------------------------------------------------------\n");

        fprintf(fp, "%d,%d,%d,%d,%.5f,%.5f,%.5f,%s\n",
            test + 1, A_rows, A_cols, B_cols, cpuTime, avgKernel, stdKernel, correct ? "PASSED" : "FAILED");

        hipFree(d_M);
        hipFree(d_N);
        hipFree(d_P);
        free(h_M);
        free(h_N);
        free(h_P);
        free(h_ref);
    }

    fclose(fp);
    printf("Results written to MP2Bonus.csv\n");
    return 0;
}
